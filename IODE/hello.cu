// includes, system
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>

// includes, project
//#include <cutil.h>

// includes, kernels
#include "hello.h"

////////////////////////////////////////////////////////////////////////////////
// declarations, forward
double * allocateDeviceAndCopy(double * hostPointer,int memsize){
    double * devicePointer;
    hipMalloc (( void **) & devicePointer , memsize);
    hipMemcpy ( devicePointer , hostPointer ,memsize, hipMemcpyHostToDevice );
    return devicePointer;
}

void generateSampleInput(int, int);
//extern "C"

#define INPUT_FMAX 2
#define INPUT_GMAX 10
////////////////////////////////////////////////////////////////////////////////
// Program main
////////////////////////////////////////////////////////////////////////////////
void generateSampleInput(int numODE,int NEQN){
    FILE* stream = fopen("sample_input.txt", "w");
    //fprintf(stream,"%d\n",NEQN);
    double value;
    for (int i=0; i<numODE; i++){
        for (int j=0; j<NEQN;j++){
            if (j < NEQN/2){
                // between 0 and 1
                value = (float) rand()/(float)(RAND_MAX);
                value*=INPUT_FMAX;
            }
            else{ 
                // between 0 and 1
                value = (float) rand()/(float)(RAND_MAX);
                // between -0.5 and 0.5
                value-=0.5;
                value*=INPUT_GMAX;
            }
            // print this equation's initial condition to file
            fprintf(stream,"%f;",value);
        }
        // start a new element
        fprintf(stream,"\n");
    }
    return;
}

int main(int argc, char** argv) {
    // number of ode systems ("elements"), e.g. 10 million
    int numODE = 10;

    // number of equations, e.g. 157
    int NEQN = 2;
    generateSampleInput(numODE,NEQN);

    // the actual equations
    double * y;
    double * g;

    //This will initiallize stuff by reading the input file.
    parseInputs(argv[1], y, g, &NEQN, &numODE);


    // double y[2];
    // double g[2];
    // y[0] = 0;
    // y[1] = 0; // cycles per second, matches spring constant

    // g[0] = 1;
    // g[1] = 2;

    double tEnd = 14;//seconds

    double t0 = 0;
    double h = 0.75;// seconds

    // Format host matrix into 1-d array
    double * yHost ;
    yHost = ( double *) malloc ( numODE * NEQN * sizeof ( double ));
    //yHost[0] = y[0];
    //yHost[1] = y[1];

    double * gHost;
    gHost = (double *) malloc ( NEQN * sizeof(double));
    //gHost[0] = g[0];
    //gHost[1] = g[1];

    /*

    for (int i = 0; i < numODE ; ++i) {
        for (int j = 0; j < NEQN ; ++j) {
            yHost [i + numODE * j] = y[i][j];
        }
    }
    */

    // allocate memory on the device and copy over
    double * yDevice ;
    yDevice = allocateDeviceAndCopy(y,numODE * NEQN * sizeof ( double ));
    double * gDevice ;
    gDevice = allocateDeviceAndCopy(g,NEQN * sizeof ( double ));

    // setup grid dimensions
    int blockSize ;
    if ( numODE < 4194304) {
        blockSize = 64;
    } 
    else if ( numODE < 8388608) {
        blockSize = 128;
    } 
    else if ( numODE < 16777216) {
        blockSize = 256;
    }
    else {
        blockSize = 512;
    }

    //printf("%d threads/block\n",blockSize);
    dim3 dimBlock ( blockSize , 1);
    //printf("%d blocks\n",numODE/dimBlock.x);
    dim3 dimGrid ( numODE / dimBlock .x+1, 1);

    // set initial time
    double t = t0;
    double tNext = t + h;
    
    //printf("before intDriver %.2f %.2f\n",g[0],g[1]);
    while (t < tEnd ) {
        // transfer memory to GPU
        if (t!=t0){
            hipMemcpy ( yDevice , yHost , numODE * NEQN * sizeof ( double ), hipMemcpyHostToDevice );
            hipMemcpy ( gDevice , gHost , NEQN * sizeof ( double ), hipMemcpyHostToDevice );
        }
        
        intDriver <<<numODE , 1 >>> (t, tNext , numODE , NEQN, gDevice , yDevice );
        
         // transfer memory back to CPU
        hipMemcpy (yHost , yDevice , numODE * NEQN * sizeof ( double ), hipMemcpyDeviceToHost );
        hipMemcpy (gHost , gDevice , NEQN * sizeof ( double ), hipMemcpyDeviceToHost );

        // for each system
        for (int j=0; j<numODE; j++){
            printf("%.4f ",t);
            for (int i=0; i<NEQN;i++){
                printf("%.4f ",yHost[i+2*j]);
        }
        printf("\n");
    }


         
        t = tNext ;
        tNext += h;
    }
    //printf("after intDriver %.2f %.2f\n",g[0],g[1]); 
    
     hipFree ( gDevice );
     hipFree ( yDevice );
}

