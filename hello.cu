// includes, system
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>

// includes, project
//#include <cutil.h>

// includes, kernels
#include "hello.h"

////////////////////////////////////////////////////////////////////////////////
// declarations, forward

//extern "C"

////////////////////////////////////////////////////////////////////////////////
// Program main
////////////////////////////////////////////////////////////////////////////////
int main(int argc, char** argv) {
    printf("Hello world\n");

    int size = 32;
    int memsize = size*sizeof(int);

    int* A;
    int* B;
    int* C;
    A = (int*) malloc(size*sizeof(int));
    B = (int*) malloc(size*sizeof(int));
    C = (int*) malloc(size*sizeof(int));

    int* d_A;
    int* d_B;
    int* d_C;

    hipMalloc((void**)&d_A, memsize);
    hipMalloc((void**)&d_B, memsize);
    hipMalloc((void**)&d_C, memsize);

    // set values of host arrays
    for (int j = 0; j<size; j++){
        A[j]=j;
        B[j]=2*j;
    }

    // copy host arrays to device
    hipMemcpy((void *)d_A,A, memsize,hipMemcpyHostToDevice);
    hipMemcpy((void *)d_B,B, memsize,hipMemcpyHostToDevice);
    hipMemcpy((void *)d_C,C, memsize,hipMemcpyHostToDevice);


    // invoke kernel
    dim3 threadsPerBlock(32,1);
    printf("Starting GPU evaluation\n");
    HelloKernel<<<1,threadsPerBlock>>>( d_A, d_B, d_C);

    // wait for threads to finish
    hipDeviceSynchronize();
    printf("\nFinished GPU evaluation\n");

    // grab output
    hipMemcpy((void *)C,(void *) d_C, memsize, 
                    hipMemcpyDeviceToHost);

    for (int i =0; i<size; i++){
        printf("%d ",C[i]);
    }
    printf("\n");

    hipFree((void *)d_A);
    hipFree((void *)d_B);
    hipFree((void *)d_C);
}

